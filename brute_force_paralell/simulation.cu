#include "hip/hip_runtime.h"

#include<stdio.h>
#include <GLFW/glfw3.h>
#include <learnopengl/shader.h>
#include<math.h>
#include <iostream>
#include <hip/hip_runtime.h>
#define N 100000
#define SCR_WIDTH 2000
#define dt 0.001
#define SCR_HEIGHT 1000
#define FACTOR 0.001
#define RADIUS 2 //Change this between 25 and 1,25 corresponds to n<=10 and 1 corresponds to n=10000
#define EPSILON 1 //Included with distance to avoid infinite acceleration
//Struct for 2-D points with two doubles as members
typedef struct points{
  double x;
  double y;
  points():x(0),y(0)
  {
  }
  points(double x1,double y1):x(x1),y(y1)
  {
  }
}point;

//Struct for storing components of 2-D velocity
typedef struct velocities{
  double vx;
  double vy;
  velocities():vx(0),vy(0)
  {
  }
  velocities(double vx1,double vy1):vx(vx1),vy(vy1)
  {
  }
}velocity;

//Struct for storing 2-D acceleration
typedef struct accelerations{
  double ax;
  double ay;
  accelerations():ax(0),ay(0)
  {
  }
  accelerations(double ax1,double ay1):ax(ax1),ay(ay1)
  {
  }
}acceleration;

point* pos = new point[N];
velocity* vel = new velocity[N];
acceleration* acc = new acceleration[N];

double distance( point curr , point aff );
void framebuffer_size_callback(GLFWwindow* window, int width, int height); //Function to change display on every window resize


__global__ void calculateForce( double* posi , acceleration* acc , velocity* vel , int *np){
        int n = *np;
        int i = threadIdx.x + blockIdx.x*blockDim.x;
        if( i < n ){
            int j;
            for ( j = 0 ; j < n ; j++){
                if( i == j )
                  continue;
                double d = sqrt((posi[2*i]-posi[2*j])*(posi[2*i]-posi[2*j])+(posi[2*i+1]-posi[2*j+1])*(posi[2*i+1]-posi[2*j+1])+EPSILON);

                if( d<4*FACTOR*RADIUS ){

                  continue;
                }
                double d3 = d*d*d;
                acc[i].ax += ( posi[2*j] - posi[2*i] )/d3;
                acc[i].ay += (  posi[2*j+1] - posi[2*i+1] )/d3;
                //Objects are too close
                //Elastic collisio
                /*
                if( d<5*FACTOR*RADIUS ){
                  //Switching velocities for collision
                  double temp;
                  temp = vel[i].vx;
                  vel[i].vx = vel[j].vx;
                  vel[j].vx = temp;
                  temp = vel[i].vy;
                  vel[i].vy = vel[j].vy;
                  vel[j].vy = temp;
                  acc[i].ax -= 2*( posi[2*j] - posi[2*i] )/d3;
                  acc[i].ay -= 2*( posi[2*j+1] - posi[2*i+1] )/d3;
                }*/

          }
          vel[i].vx += acc[i].ax*dt;
          posi[2*i] += vel[i].vx*dt;
          vel[i].vy += acc[i].ay*dt;
          posi[2*i+1] += vel[i].vy*dt;
          acc[i].ax = 0;
          acc[i].ay = 0;
        }
  }

int main( void ){

    int i,n,j,*dev_n;
    double *dev_pos;
    acceleration *dev_acc;
    velocity *dev_vel;
    std::cout<<"Enter the number of particles:"<<std::endl;
    std::cin>>n;
    //Initiating the window to draw
    GLFWwindow* window;
    //Initiating the GLFW window
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    #ifdef __APPLE__
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE); // uncomment this statement to fix compilation on OS X
  #endif
    window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
    glfwMakeContextCurrent(window);
    //Creating a viewport
    glViewport(0, 0, SCR_WIDTH, SCR_HEIGHT);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);


    //For loop generating random doubles which constitutes the position of the particles
    for( i = 0 ; i < n ; i++ ){

        pos[i].x = 2*(float)rand()/(float)(RAND_MAX)-1.1;

        if( pos[i].x < -0.9){
            pos[i].x += 0.2;
        }

        pos[i].y = 2*(float)rand()/(float)(RAND_MAX)-1.1;

        if( pos[i].y < -0.9){
            pos[i].y += 0.2;
        }
        //Prevent intersection of elements at initialization
        //Delete this for large number of paricles
        for( j = 0 ; j < i ; j++ ){
            double d = distance(pos[i],pos[j]);
            if( d < 2*FACTOR*RADIUS ){
               i--;
               break;
            }
        }
      }


    hipMalloc((void**)&dev_pos,N*sizeof(point));
    hipMalloc((void**)&dev_vel,N*sizeof(point));
    hipMalloc((void**)&dev_acc,N*sizeof(point));
    hipMalloc((void**)&dev_n,sizeof(int));
    hipMemcpy(dev_n,&n,sizeof(int),hipMemcpyHostToDevice);

    //Creating a vertex array object
    unsigned int VAO;
    glGenVertexArrays(1, &VAO);

    //Creating a vertex buffer object
    unsigned int VBO;
    glGenBuffers(1,&VBO);
    glBindBuffer(GL_ARRAY_BUFFER, VBO);

    double posi[2*n];
    for( i = 0 ; i < n ; i++){
        posi[2*i] = pos[i].x;
        posi[2*i+1] = pos[i].y;
    }
    //Copying the data into gpu's memory
    glBufferData(GL_ARRAY_BUFFER, sizeof(posi), posi, GL_DYNAMIC_DRAW);
    glBindVertexArray(VAO);


    //Creatiing a vertex attribute pointer for the points
    glVertexAttribPointer(0, 2, GL_DOUBLE, GL_FALSE, 2*sizeof(double), (void*)(0));
    glEnableVertexAttribArray(0);

    Shader shader("vertex_shader_source", "fragment_shader_source");
    hipMemcpy(dev_pos,posi,2*n*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_vel,vel,n*sizeof(velocity),hipMemcpyHostToDevice);
    hipMemcpy(dev_acc,acc,n*sizeof(acceleration),hipMemcpyHostToDevice);
    glEnable(GL_POINT_SMOOTH);
    glEnable(GL_BLEND);
    glEnable( GL_POINT_SPRITE );
    while (!glfwWindowShouldClose(window))
    {

        glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        shader.use();
        //Copying the data into gpu's memory
        glBindBuffer(GL_ARRAY_BUFFER, VBO);
        glBufferData(GL_ARRAY_BUFFER, sizeof(posi), posi, GL_DYNAMIC_DRAW);
        glBindVertexArray(VAO);
        glPointSize(2*RADIUS);
        glDrawArrays(GL_POINTS,0,n);
        glfwSwapBuffers(window);
        glfwPollEvents();
        calculateForce<<<65535,1024>>>(dev_pos,dev_acc,dev_vel,dev_n);  //For larger values of n change this to 63353 instead of (n+1023)/1024
        hipMemcpy(posi,dev_pos,2*n*sizeof(double),hipMemcpyDeviceToHost);

      }
      glfwTerminate();
      return 0;
}


double distance( point curr , point aff ){
    return sqrt((curr.x-aff.x)*(curr.x-aff.x)+(curr.y-aff.y)*(curr.y-aff.y));
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
  {
      glViewport(0, 0, width, height);
  }
